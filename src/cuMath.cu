#include "hip/hip_runtime.h"
#include "cuMath.h"
static int MAX_THREADNUM = Devices::instance()->max_ThreadsPerBlock();
__global__ void ReLU_kernel(float* src,
		float* dst,//目的
		int rows,//行数
		int cols,//列数
		int maxt)//每个线程块允许的最大线程数
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		assert(x * cols + y < rows * cols);
		if (src[x * cols + y] <= 0)
		{
			dst[x * cols + y] = 0;
		}
		else
		{
			dst[x * cols + y] = src[x * cols + y];
		}
		y += maxt;
	}
}

cuMatrix ReLU(cuMatrix& cumat)//线性修正
{
	cuMatrix res(cumat.rows(), cumat.cols());//大小不变的矩阵
	int threadnum = MAX_THREADNUM > cumat.cols() ? cumat.cols() : MAX_THREADNUM;
	dim3 blocks=dim3(cumat.rows());
	dim3 threads=dim3(threadnum);
	ReLU_kernel<<<blocks, threads>>>(cumat.getDev(),
			res.getDev(), cumat.rows(), cumat.cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("ReLU ");
	return res;
}

__global__ void dReLU_kernel(float* src,
		float* dst,
		int cols,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		if (src[x * cols + y] <= 0)
		{
			dst[x * cols + y] = 0;
		}
		else
		{
			dst[x * cols + y] = 1;
		}
		y += maxt;
	}
}

cuMatrix dReLU(cuMatrix& cumat)
{
	cuMatrix res(cumat.rows(), cumat.cols());
	int threadnum = MAX_THREADNUM > cumat.cols() ? cumat.cols() : MAX_THREADNUM;
	dim3 blocks=dim3(cumat.rows());
	dim3 threads=dim3(threadnum);
	dReLU_kernel<<<blocks,threads>>>(cumat.getDev(),
			res.getDev(), cumat.cols(), MAX_THREADNUM);
	getLastCudaError("dReLU ");
	return res;
}

__global__ void reduce_max_kernel(float* dev_x,
		float* dev_y,//存放结果
        int rows,//行
		int cols, //列
		int maxt)//每个线程块的最大线程数
{
	int tid = threadIdx.x;//因为只有一个线程块
	while (tid < cols)
	{
		float max = (float) LONG_MIN;//长整形最小值
		for (int i = 0; i < rows; i++)
		{
			max = max > dev_x[i * cols + tid] ? max : dev_x[i * cols + tid];
		}
		for (int i = 0; i < rows; i++)
		{
			dev_y[i * cols + tid] = max;
		}
		tid += maxt;
	}
}

cuMatrix reduceMax(cuMatrix src)
{
	cuMatrix res(src.rows(), src.cols());
	int threadnum = MAX_THREADNUM > src.cols() ? src.cols() : MAX_THREADNUM;
	dim3 blocks=dim3(1);
	dim3 threads=dim3(threadnum);
	reduce_max_kernel<<<blocks,threads>>>(src.getDev(), res.getDev(),
			src.rows(), src.cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("reduce_max");
	return res;
}

//  share memory?
__global__ void reduce_sum_kernel(float* dev_x,
		float* dev_y,
		int rows,
		int cols,
		int maxt)
{
	int tidx = blockIdx.x;
	int tidy = threadIdx.x;
	float sum = 0;//每列之和
	while (tidy < cols)
	{
		for (int i = 0; i < rows; i++)
		{
			sum += dev_x[i * cols + tidy];
		}
		dev_y[tidx * cols + tidy] = sum;
		tidy += maxt;
	}
}


cuMatrix reduceSum(cuMatrix src)
{
	cuMatrix res(src.rows(), src.cols());
	int threadnum = MAX_THREADNUM > src.cols() ? src.cols() : MAX_THREADNUM;
	dim3 blocks=dim3(src.rows());
	dim3 threads=dim3(threadnum);
	reduce_sum_kernel<<<blocks, threads>>>(src.getDev(),
			res.getDev(), src.rows(), src.cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("reduce_sum");
	return res;
}

//行做block
__global__ void log_kernel(float* dev_x,
		float* dev_y,
		int cols,
		int maxt)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	while (tid < cols)
	{
		dev_y[bid * cols + tid] = log(dev_x[bid * cols + tid]);
		tid += maxt;
	}
}

//矩阵中每个值取log
cuMatrix Log(cuMatrix src)
{
	cuMatrix res(src.rows(), src.cols());
	int threadnum = MAX_THREADNUM > src.cols() ? src.cols() : MAX_THREADNUM;
	dim3 blocks=dim3(src.rows());
	dim3 threads=dim3(threadnum);
	log_kernel<<<blocks, threads>>>(src.getDev(),
			res.getDev(), src.cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("ElementLog");

	return res;
}

//每个元素取指数
__global__ void exp_kernel(float* dev_x,
		float* dev_y,
		int cols,
		int maxt)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	while (tid < cols)
	{
		//取每个数e为底的指数
		dev_y[bid * cols + tid] = exp(dev_x[bid * cols + tid]);
		tid += maxt;
	}
}

//e为底的指数次方
cuMatrix Exp(cuMatrix src)
{
	cuMatrix res(src.rows(), src.cols());
	int threadnum = MAX_THREADNUM > src.cols() ? src.cols() : MAX_THREADNUM;
	dim3 blocks=dim3(src.rows());
	dim3 threads=dim3(threadnum);
	exp_kernel<<<blocks,threads>>>(src.getDev(),
			res.getDev(),
			src.cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("ElementExp");
	return res;
}

__global__ void Pow_kernel(float* dev_x,
		float* dev_y,
		float* dev_z,//存放结果
		int cols,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		dev_z[x * cols + y] = pow(dev_x[x * cols + y], dev_y[x * cols + y]);
		y += maxt;
	}
}

//x矩阵中每个元素y次方  核函数
__global__ void Pow_kernel(float* dev_x,
		float y_,
		float* dev_z,
		int cols,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		dev_z[x * cols + y] = pow(dev_x[x * cols + y], y_);
		y += maxt;
	}
}

//x矩阵中每个元素y矩阵中对应元素次方
cuMatrix Pow(cuMatrix x,cuMatrix y)
{
	if (!(x.rows() == y.rows()))
	{
		printf("cuMatrix Pow(cuMatrix x,cuMatrix y) error: rows!\n");
		exit(0);
	}
	if (!(x.cols() == y.cols()))
	{
		printf("cuMatrix Pow(cuMatrix x,cuMatrix y) error: cols!\n");
		exit(0);
	}
	cuMatrix res(x.rows(), x.cols());
	int threadnum = MAX_THREADNUM > x.cols() ? x.cols() : MAX_THREADNUM;
	dim3 blocks=dim3(x.rows());
	dim3 threads=dim3(threadnum);
	Pow_kernel<<<blocks,threads>>>(x.getDev(),
			y.getDev(), res.getDev(), x.cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("cuMatrix Pow(cuMatrix x,cuMatrix y)");
	return res;
}

//x矩阵中每个元素y次方
cuMatrix Pow(cuMatrix x,float y)
{
	int threadnum = MAX_THREADNUM > x.cols() ? x.cols() : MAX_THREADNUM;
	cuMatrix res(x.rows(), x.cols());
	dim3 blocks=dim3(x.rows());
	dim3 threads=dim3(threadnum);
	Pow_kernel<<<blocks,threads>>>(x.getDev(), y,
			res.getDev(), x.cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("ElementPow matrix float matrix ");
	return res;
}
