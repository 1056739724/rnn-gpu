#include "hip/hip_runtime.h"
#include "cuMatrix.h"
#include "hardware.h"


static int MAX_THREADNUM = Devices::instance()->max_ThreadsPerBlock();

//hipblasHandle_t类型是一个不透明指针类型结构支持CUBLAS库内容。
//CUBLAS库内容必须通过hipblasCreate()初始化，
//返回的处理结果必须经过所有后续库函数调用，最后必须使用hipblasDestroy()摧毁其中内容
hipblasHandle_t& getHandle()
{
	static hipblasHandle_t handle = NULL;
	if (handle == NULL)
	{
		hipblasStatus_t status;
		status = hipblasCreate(&handle);
		if (status != HIPBLAS_STATUS_SUCCESS)
		{
			printf("init: CUBLAS initialization failed\n");
			exit(0);
		}
	}
	return handle;
}

//两个矩阵相加核函数
__global__ void add_kernel(float* dev_x,
		float* dev_y,
		float* dev_z,//存放结果
		int cols,
		int maxt)//每个线程块线程的最大数量
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		dev_z[x * cols + y] = dev_x[x * cols + y] + dev_y[x * cols + y];
		y += maxt;
	}
}

//矩阵加上一个float类型的数
__global__ void add_kernel(float* dev_x,
		float y_,
		float* dev_z,//存放结果
		int cols,
		int maxt)//每个线程块线程的最大数量
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		dev_z[x * cols + y] = dev_x[x * cols + y] + y_;
		y += maxt;
	}
}

cuMatrix cuMatrix::operator +(cuMatrix cumat)//两个矩阵相加，+号运算符重载
{
	if (!size)
	{
		if (cumat.data->getDev() == NULL)
		{
			printf("cuMatrix error : both matrix are empty.\n");
			exit(0);
		}
		cuMatrix res = cumat;
		return res;
	}
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res(rows(), cols());
	dim3 blocks=dim3(rows());
	dim3 threads=dim3(threadnum);
	add_kernel<<<blocks,threads>>>(data->getDev(),
			cumat.data->getDev(), res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pre-element add cuMatrix + cuMatrix");
	return res;
}

cuMatrix cuMatrix::operator +(float i)
{
	assert(data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res(rows(), cols());
	add_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pre-element add cuMatrix + float");
	return res;
}

//矩阵与矩阵相减
__global__ void dec_kernel(float* dev_x,
		float* dev_y,
		float* dev_z, //存放运算结果
		int cols,//列
		int maxt)//每块最大线程数量
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		dev_z[x * cols + y] = dev_x[x * cols + y] - dev_y[x * cols + y];
		y += maxt;
	}
}

//矩阵减去一个数
__global__ void dec_kernel(float* dev_x,
		float y_,
		float* dev_z,
		int cols,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		dev_z[x * cols + y] = dev_x[x * cols + y] - y_;
		y += maxt;
	}
}

cuMatrix cuMatrix::operator -(cuMatrix cumat)//两个矩阵相减
{
	if (!size)
	{
		if (cumat.data->getDev() == NULL)
		{
			printf("cuMatrix error : both matrix are empty.\n");
			exit(0);
		}
		cuMatrix res = cumat * -1.0f;
		return res;
	}
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res(rows(), cols());
	dim3 blocks=dim3(rows());
	dim3 threads=dim3(threadnum);
	dec_kernel<<<blocks,threads>>>(data->getDev(),
			cumat.data->getDev(), res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pre-element add cuMatrix - cuMatrix");
	return res;
}

cuMatrix cuMatrix::operator -(float i)
{
	assert(data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res(rows(), cols());
	dec_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pre-element add cuMatrix - float");
	return res;
}

__global__ void mul_kernel(float* dev_x,
		float* dev_y,
		float* dev_z,
		int cols,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols) {
		dev_z[x * cols + y] = dev_x[x * cols + y] * dev_y[x * cols + y];
		y += maxt;
	}
}

__global__ void mul_kernel(float* dev_x,
		float y_,
		float* dev_z,
		int cols,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		dev_z[x * cols + y] = dev_x[x * cols + y] * y_;
		y += maxt;
	}
}

//矩阵相乘，对应元素相乘
cuMatrix cuMatrix::Mul(cuMatrix cumat)
{
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res(rows(), cols());
	dim3 blocks=dim3(rows());
	dim3 threads=dim3(threadnum);
	mul_kernel<<<blocks, threads>>>(data->getDev(),
			cumat.data->getDev(), res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pre-element add cuMatrix * cuMatrix");
	return res;
}

//矩阵中每个元素乘以一个数
cuMatrix cuMatrix::operator *(float i)
{
	assert(data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res(rows(), cols());
	mul_kernel<<<dim3(rows()), dim3(threadnum)>>>(data->getDev(), i,
			res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pre-element add cuMatrix * float");
	return res;
}
//res = this * cumat

//res = this * cumat
cuMatrix cuMatrix::operator *(cuMatrix cumat)
{
	//assert宏的原型定义在<assert.h>中，其作用是如果它的条件返回错误，则终止程序执行，原型定义：
	//assert的作用是现计算表达式 expression ，如果其值为假（即为0），
	//那么它先向stderr打印一条出错信息，然后通过调用 abort 来终止程序运行
	//http://www.cnblogs.com/ggzss/archive/2011/08/18/2145017.html
	assert(cols() == cumat.rows());//1255
	cuMatrix res(rows(), cumat.cols());
	float alpha = 1.0;
	float beta = 0.0;
	hipblasStatus_t stat;
	stat = hipblasSgemm(getHandle(),
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			cumat.cols(),
			rows(),
			cumat.rows(),
			&alpha,
			cumat.getDev(),
			cumat.cols(),
			getDev(),
			cols(),
			&beta,
			res.getDev(),
			res.cols());
	hipStreamSynchronize(0);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf("cuMatrix::Mul() error\n");
		exit(0);
	}
	return res;
}

__global__ void div_kernel(float* dev_x,
		float* dev_y,
		float* dev_z,//存放结果
		int cols,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		if (dev_y[x * cols + y] != 0)
		{
			dev_z[x * cols + y] = dev_x[x * cols + y] / dev_y[x * cols + y];
		}
		y += maxt;
	}
}

__global__ void div_kernel(float* dev_x,
		float y_,//矩阵中每个元素将要除以y_
		float* dev_z, //存放结果
		int cols,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		if (y_ != 0)
		{
			dev_z[x * cols + y] = dev_x[x * cols + y] / y_;
		}
		y += maxt;
	}
}

//应该是对应元素相除
cuMatrix cuMatrix::operator /(cuMatrix cumat)
{
	assert(cumat.rows() == rows() && cumat.cols() == cols());
	assert(data->getDev() != NULL && cumat.data->getDev() != NULL);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res(rows(), cols());
	dim3 blocks=dim3(rows());
	dim3 threads=dim3(threadnum);
	div_kernel<<<blocks, threads>>>(data->getDev(),
			cumat.data->getDev(), res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pre-element add cuMatrix / cuMatrix");
	return res;
}

//矩阵中每个元素除以i
cuMatrix cuMatrix::operator /(float i)
{
	assert(data->getDev() != NULL);
	assert(i != 0);
	int threadnum = MAX_THREADNUM > cols() ? cols() : MAX_THREADNUM;
	cuMatrix res(rows(), cols());
	dim3 blocks=dim3(rows());
	dim3 threads=dim3(threadnum);
	div_kernel<<<blocks,threads>>>(data->getDev(), i,
			res.data->getDev(), cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pre-element add cuMatrix / float");
	return res;
}

__global__ void t_kernel(float* dev_src, float* dev_res, int res_r, int res_c,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < res_c)
	{
		dev_res[x * res_c + y] = dev_src[y * res_r + x];
		y += maxt;
	}
}

cuMatrix cuMatrix::t()
{
	assert(cols() != 0 && rows() != 0);
	cuMatrix res(cols(), rows());
	int threadnum = MAX_THREADNUM > res.cols() ? res.cols() : MAX_THREADNUM;
	t_kernel<<<dim3(res.rows()), dim3(threadnum)>>>(data->getDev(),
			res.data->getDev(), res.rows(), res.cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("pre-element add cuMatrix / float");
	return res;
}

__global__ void Div_kernel(float x_,
		float* dev_y,
		float* dev_z,
		int cols,
		int maxt)
{
	int x = blockIdx.x;
	int y = threadIdx.x;
	while (y < cols)
	{
		if (dev_y[x * cols + y] != 0)
		{
			dev_z[x * cols + y] = x_ / dev_y[x * cols + y];
		}
		y += maxt;
	}
}
cuMatrix operator /(float x, cuMatrix cumat)
{
	cuMatrix res(cumat.rows(), cumat.cols());
	int threadnum = MAX_THREADNUM > cumat.cols() ? cumat.cols() : MAX_THREADNUM;
	Div_kernel<<<dim3(cumat.rows()), dim3(threadnum)>>>(x, cumat.getDev(),
			res.getDev(), cumat.cols(), MAX_THREADNUM);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("ElementDiv double matrix matrix ");
	return res;
}


