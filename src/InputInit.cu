#include "hip/hip_runtime.h"
#include "InputInit.h"


__global__ void set_sampleY_kernel(float* sampleY,//5*50训练集标签
		int* src, //训练集标签数据
		int* dev_ran,//0到49的打乱次序
		int cols, //50
		int ngram) //5
{
	int tid = threadIdx.x;//0——4
	int bid = blockIdx.x;//0-——49
	sampleY[tid * cols + bid] = src[dev_ran[bid] * ngram + tid];
}


__global__ void set_acti0_kernel(float** acti0,
		//src：存放原始trainx：vector<vector<int>>，即原始数据第一列对应的从0开始到1254的一个值
		int* src,//已经上传到gpu上的训练集
		int* dev_ran,//0到49的打乱次序
		int cols,//50
		int ngram)
{//5个线程，50个线程块
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	float *p = acti0[tid];//第tid个mat的地址，acti0原本有5个矩阵，每个矩阵1255*50
	//src的维数有trainX.size()个，每一维5个int大小
	int n = src[dev_ran[bid] * ngram + tid];//dev_ran[bid] * ngram + tid：每次得到的是每个子句子的每一行
	//n是0到1254的一个值   因为矩阵转为指针都是按行存放的故下面式子正确
	p[n * cols + bid] = 1;//p指向某一个矩阵，故p[0]、p[1]....p[1255]，这是给矩阵第n*cols行第bid列赋值
}


void init_acti0(cuMatrixVector& acti_0,//vector<mat>，里面存放5个矩阵，每个矩阵1255*50
		cuMatrix& sampleY,//mat 5*50  存放50个子句子，每个子句子5行的类别
		int nGram,//每个子句子5行
		int batch_Size)//随机取50个子句子
{
	int batch_size = batch_Size;//50
	int ngram = nGram;//5

	int *dev_ran = NULL;//存放打乱之后的数字，且在gpu上面
	Samples::instance()->randproductor_init();//初始化randproductor：int*  存放从0到train_num（训练集子句子个数）
	hipError_t cudaStat = hipMalloc((void**) &dev_ran, batch_size * sizeof(int));
	if (cudaStat != hipSuccess)
	{
		printf("init_acti0 failed\n");
		exit(0);
	}

	checkCudaErrors(hipMemcpy(dev_ran,
			        Samples::instance()->get_rand(1),//打乱数据之后
					batch_size * sizeof(int),//拷贝大小50个数
					hipMemcpyHostToDevice));
	//内建dim3类型,是一个三维数组，可以用于指定启动的线程块的数量：定义grid和block的组织方法。
	dim3 block = dim3(batch_size);//50
	dim3 thread = dim3(ngram);//5
    //<<<>>>中第一个参数表示设备在执行核函数时使用的并行线程块的数量
	//第二个参数表示cuda运行时在每个线程块中创建的线程数量
	set_acti0_kernel<<<block, thread>>>(acti_0.get_devPoint(),//已经在gpu上面
			Samples::instance()->get_trainX(),//gpu上的训练集
			dev_ran,//50个打乱的子句子的序号
			batch_size, //50
			ngram);//5

    //涉及到多个内核函数运行的时候就需要了，内核函数中各个线程在运行的时候不是同步的，所以在计算完成的时候一般需要同步一下；
	checkCudaErrors(hipDeviceSynchronize());//同步
	getLastCudaError("set_acti0_kernel-2");

	set_sampleY_kernel<<<block, thread>>>(sampleY.getDev(),
			Samples::instance()->get_trainY(), //gpu上的训练集标签
			dev_ran, //0到49的打乱次序
			batch_size, //50
			ngram);//5
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("set_sampleY_kernel-2");
	checkCudaErrors(hipFree(dev_ran));
}


__global__ void set_gt_kernel(float** gt_, //将要存放类别的向量
		float* y,
		int rows, //5
		int cols)//50
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	assert(tid < rows && bid < cols);
	float* p = gt_[tid];//gt_中有5个矩阵，每个矩阵11*50
	int i = y[tid * cols + bid];//这个i的值应该是类别
	assert(i < 12);
	p[i * cols + bid] = 1.0;
}

//把真实分类类别放到gt中，gt已经在gpu上面
void set_groundtruth(cuMatrixVector& gt, cuMatrix& sampleY)
{
	//sampleY:5*50
	dim3 block = dim3(sampleY.cols());//50
	dim3 thread = dim3(sampleY.rows());//5
	set_gt_kernel<<<block, thread>>>(gt.get_devPoint(), sampleY.getDev(),
			sampleY.rows(), sampleY.cols());
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("set_groundtruth ");
}


void initTestdata(vector<vector<int> > &testX,
		vector<vector<int> > &testY,
		int nGram)
{
	//为两个指针分配空间
	int *host_X = (int *) malloc(sizeof(int) * testX.size() * nGram);
	int *host_Y = (int *) malloc(sizeof(int) * testY.size() * nGram);

	for (int i = 0; i < testX.size(); i++)//测试集子句子的个数，每个子句子5行
	{
		//每个子句子的第一行开始，5行大小，拷贝进去，再拷下一个子句子的5行
		memcpy(host_X + i * nGram, &testX[i][0], sizeof(int) * nGram);
	}
	for (int i = 0; i < testY.size(); i++)
	{
		memcpy(host_Y + i * nGram, &testY[i][0], sizeof(int) * nGram);
	}
	//数据上传至gpu上
	Samples::instance()->testX2gpu(host_X, sizeof(int) * testX.size() * nGram);
	Samples::instance()->testY2gpu(host_Y, sizeof(int) * testY.size() * nGram);
	free (host_X);
	free (host_Y);
}


void initTraindata(vector<vector<int> > &trainX,
		vector<vector<int> > &trainY,
		int nGram)
{
	//trainX占用空间sizeof(int) * trainX.size() * nGram
	int *host_X = (int *) malloc(sizeof(int) * trainX.size() * nGram);
	int *host_Y = (int *) malloc(sizeof(int) * trainY.size() * nGram);
	//数据拷贝
	for (int i = 0; i < trainX.size(); i++)
	{
		//void*memcpy(void*dest, const void*src,unsigned int count);
		memcpy(host_X + i * nGram, &trainX[i][0], sizeof(int) * nGram);
	}
	for (int i = 0; i < trainY.size(); i++)
	{
		memcpy(host_Y + i *nGram, &trainY[i][0], sizeof(int) * nGram);
	}
	//训练集数据上传到gpu
	Samples::instance()->trainX2gpu(host_X,sizeof(int) * trainX.size() * nGram);
	Samples::instance()->trainY2gpu(host_Y,sizeof(int) * trainY.size() * nGram);
	free (host_X);
	free (host_Y);
}


void Data2GPU(vector<vector<int> > &trainX,//原始数据第一列对应的int值
		vector<vector<int> > &trainY,//原始数据第二列得到的int值
		vector<vector<int> > &testX,
		vector<vector<int> > &testY,
		int nGram)
{
	initTraindata(trainX,trainY,nGram);
	initTestdata(testX,testY,nGram);

}


__global__ void getDataMat_kernel(float** sampleX, int* src, int off, int cols,
		int ngram)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	float *p = sampleX[tid];
	int n = src[(off + bid) * ngram + tid];
	p[n * cols + bid] = 1.0;
}

void getDataMat(cuMatrixVector &sampleX, int off, int bs, int n,
		bool flag,int ngram)
{
		int n_gram = ngram;
		for (int i = 0; i < 5; i++)
		{
			sampleX.push_back(new cuMatrix(n, bs));
		}
		sampleX.toGpu();
		dim3 thread = dim3(n_gram);
		dim3 block = dim3(bs);
		if (flag)
		{
			getDataMat_kernel<<<block, thread>>>(sampleX.get_devPoint(),
					Samples::instance()->get_trainX(), off, bs, n_gram);
		}
		else
		{
			getDataMat_kernel<<<block, thread>>>(sampleX.get_devPoint(),
					Samples::instance()->get_testX(), off, bs, n_gram);
		}
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("getDataMat_kernel ");

}

__global__ void get_res_array_kernel(float* src, int* dev_res, int rows,
		int cols) {
	int bid = blockIdx.x;
	float max = src[bid];
	dev_res[bid] = 0;
	for (int i = 1; i < rows; i++) {
		if (max < src[i * cols + bid]) {
			max = src[i * cols + bid];
			dev_res[bid] = i;
		}
	}
}

void get_res_array(cuMatrix src, int *res, int offset)
{
	int *dev_res;
	checkCudaErrors(hipMalloc((void** )&dev_res, sizeof(int) * src.cols()));
	get_res_array_kernel<<<src.cols(), 1>>>(src.getDev(), dev_res, src.rows(),
			src.cols());
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("get_res_array ");
	checkCudaErrors(
			hipMemcpy(res + offset, dev_res, sizeof(int) * src.cols(),
					hipMemcpyDeviceToHost));
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(dev_res));
}

__global__ void set_label_kernel(int* dst, int *src, int num, int threadnum,
		int mid) {
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int off = bid * threadnum + tid;
	if (off < num) {
		dst[off] = src[off * (mid * 2 + 1) + mid];
	}
}

void set_label(int* label, int size,bool flag)
{
	int *dev_label;
	int mid =5/ 2;
	int num = size;
	checkCudaErrors(hipMalloc((void** )&dev_label, sizeof(int) * num));
	int threadnum = Devices::instance()->max_ThreadsPerBlock() > num ? num : Devices::instance()->max_ThreadsPerBlock();
	int blocknum = num / threadnum + 1;
	dim3 blocks(blocknum);
	dim3 threads(threadnum);
	if (flag) {
		set_label_kernel<<<blocks, threads>>>(dev_label,
				Samples::instance()->get_trainY(), num, threadnum, mid);
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("set_label");
	} else {
		set_label_kernel<<<blocks, threads>>>(dev_label,
				Samples::instance()->get_testY(), num, threadnum, mid);
		checkCudaErrors(hipDeviceSynchronize());
		getLastCudaError("set_label");
	}
	checkCudaErrors(
			hipMemcpy(label, dev_label, sizeof(int) * num,
					hipMemcpyDeviceToHost));
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipFree(dev_label));
	getLastCudaError("set_label2");
}

